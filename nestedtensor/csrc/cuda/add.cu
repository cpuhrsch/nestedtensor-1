#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <nestedtensor/csrc/cuda/add.h>
#include <stdio.h>

namespace nested_tensor {
namespace cuda {

__global__
void add_scalars(
    c10::Half* input,
    c10::Half* scalars,
    c10::Half* output,
    const int input_outer_stride,
    const int* offsets)
{
  const int batch_id  = blockIdx.x;
  const int scalars_id  = batch_id / input_outer_stride;
  const int grain_size = blockDim.x;
  const int tid = threadIdx.x;
  const int range = (offsets[batch_id + 1] - offsets[batch_id]);
  const int num_chunks = range / grain_size;
  for (int id = 0; id < num_chunks; id++) {
    output[offsets[batch_id] + id * grain_size + tid] =
      input[offsets[batch_id] + id * grain_size + tid] + scalars[scalars_id];
  }
  const int leftover = num_chunks * grain_size;
  if (leftover + tid < range) {
    output[offsets[batch_id] + leftover + tid] =
      input[offsets[batch_id] + leftover + tid] + scalars[scalars_id];
  }
}

void add_scalar_kernelLauncher(
    c10::Half* input, // [batch_size x offsets[-1]]
    c10::Half* scalars, // [batch_size]
    c10::Half* output, // [batch_size x offsets[-1]]
    const int batch_size,
    const int input_outer_stride,
    const int* offsets /* [batch_size] */,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;

  add_scalars<<<grid, 256, 0, stream>>>(
      input,
      scalars,
      output,
      input_outer_stride,
      offsets);
}

__global__
void mul_scalars(
    c10::Half* input,
    c10::Half* scalars,
    c10::Half* output,
    const int input_outer_stride,
    const int* offsets)
{
  const int batch_id  = blockIdx.x;
  const int scalars_id  = batch_id / input_outer_stride;
  const int grain_size = blockDim.x;
  const int tid = threadIdx.x;
  const int range = (offsets[batch_id + 1] - offsets[batch_id]);
  const int num_chunks = range / grain_size;
  for (int id = 0; id < num_chunks; id++) {
    output[offsets[batch_id] + id * grain_size + tid] =
      input[offsets[batch_id] + id * grain_size + tid] * scalars[scalars_id];
  }
  const int leftover = num_chunks * grain_size;
  if (leftover + tid < range) {
    output[offsets[batch_id] + leftover + tid] =
      input[offsets[batch_id] + leftover + tid] * scalars[scalars_id];
  }
}

void mul_scalar_kernelLauncher(
    c10::Half* input, // [batch_size x offsets[-1]]
    c10::Half* scalars, // [batch_size]
    c10::Half* output, // [batch_size x offsets[-1]]
    const int batch_size,
    const int input_outer_stride,
    const int* offsets /* [batch_size] */,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;

  mul_scalars<<<grid, 256, 0, stream>>>(
      input,
      scalars,
      output,
      input_outer_stride,
      offsets);
}

__global__
void sub_scalars(
    c10::Half* input,
    c10::Half* scalars,
    c10::Half* output,
    const int input_outer_stride,
    const int* offsets)
{
  const int batch_id  = blockIdx.x;
  const int scalars_id  = batch_id / input_outer_stride;
  const int grain_size = blockDim.x;
  const int tid = threadIdx.x;
  const int range = (offsets[batch_id + 1] - offsets[batch_id]);
  const int num_chunks = range / grain_size;
  for (int id = 0; id < num_chunks; id++) {
    output[offsets[batch_id] + id * grain_size + tid] =
      input[offsets[batch_id] + id * grain_size + tid] - scalars[scalars_id];
  }
  const int leftover = num_chunks * grain_size;
  if (leftover + tid < range) {
    output[offsets[batch_id] + leftover + tid] =
      input[offsets[batch_id] + leftover + tid] - scalars[scalars_id];
  }
}

void sub_scalar_kernelLauncher(
    c10::Half* input, // [batch_size x offsets[-1]]
    c10::Half* scalars, // [batch_size]
    c10::Half* output, // [batch_size x offsets[-1]]
    const int batch_size,
    const int input_outer_stride,
    const int* offsets /* [batch_size] */,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;

  sub_scalars<<<grid, 256, 0, stream>>>(
      input,
      scalars,
      output,
      input_outer_stride,
      offsets);
}

__global__
void batchnorm_inference(
    const c10::Half* input,
    const c10::Half* mean,
    const c10::Half* running_var,
    const c10::Half eps,
    const c10::Half* weight,
    const c10::Half* bias,
    c10::Half* output,
    const int input_outer_stride,
    const int* offsets)
{
  const int batch_id  = blockIdx.x;
  const int grid_id  = blockIdx.y;
  const int scalars_id  = batch_id / input_outer_stride;
  const int grain_size = 256 * 2;
  const int tid = threadIdx.x + grid_id * 256;
  const int range = (offsets[batch_id + 1] - offsets[batch_id]);
  const int num_chunks = range / grain_size;
  c10::Half value = running_var[scalars_id] + eps;
  value = hrsqrt(value);
  value = value * weight[scalars_id];
  c10::Half value2 = mean[scalars_id] * value - bias[scalars_id];

  int input_offset = offsets[batch_id] + tid;
  int id = 0;
  for (; id < num_chunks; id++) {
    output[input_offset] = __ldg(reinterpret_cast<const __half*>(input) + input_offset) * value - value2;
    input_offset += grain_size;
  }
  if (input_offset < offsets[batch_id + 1]) { //leftover + tid < range) {
    output[input_offset] = __ldg(reinterpret_cast<const __half*>(input) + input_offset) * value - value2;
  }
}

void batchnorm_inference_kernelLauncher(
    c10::Half* input, // [batch_size x offsets[-1]]
    c10::Half* mean, // [batch_size]
    c10::Half* running_var,
    c10::Half eps,
    c10::Half* weight, // [batch_size]
    c10::Half* bias, // [batch_size]
    c10::Half* output, // [batch_size x offsets[-1]]
    const int batch_size,
    const int input_outer_stride,
    const int* offsets /* [batch_size] */,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;
  grid.y = 2;

  batchnorm_inference<<<grid, 256, 0, stream>>>(
      input,
      mean,
      running_var,
      eps,
      weight,
      bias,
      output,
      input_outer_stride,
      offsets);
}

}
}
