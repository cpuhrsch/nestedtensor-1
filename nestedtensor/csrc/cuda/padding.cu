#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <nestedtensor/csrc/cuda/attention.h>
#include <stdio.h>

namespace nested_tensor {
namespace cuda {

template<typename T>
__global__
void add_padding(
    const T* input,
    T* output,
    const int* offsets,
    const int batch_size,
    const int output_stride,
    const int inner_size) 
{
  const int batch_id  = blockIdx.x;
  const int grain_size = blockDim.x;
  const int tid = threadIdx.x;
  const int range = (offsets[batch_id + 1] - offsets[batch_id]) * inner_size;
  const int num_chunks = range / grain_size;
  for (int id = 0; id < num_chunks; id++) {
    output[batch_id * output_stride + id * grain_size + tid]
      = input[offsets[batch_id] * inner_size + id * grain_size + tid];
  }
  const int leftover = num_chunks * grain_size;
  if (leftover + tid < range) {
    output[batch_id * output_stride + leftover + tid]
      = input[offsets[batch_id] * inner_size + leftover + tid];
  }
}

template<typename T>
void add_padding_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    const int* offsets, // [batch_size]
    const int batch_size,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;

  add_padding<float><<<grid, 1024, 0, stream>>>(
      input,
      output,
      offsets,
      batch_size,
      output_stride,
      inner_size);
}

template void add_padding_kernelLauncher<float>(
    float* input,
    float* output,
    const int* offsets,
    const int batch_size,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream);

template<typename T>
__global__
void add_padding_mask(
    const T* input,
    T* output,
    int* output_mask,
    const int* offsets,
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size)
{
  const int batch_id  = blockIdx.x;
  for (int i = 0; i < (offsets[batch_id + 1] - offsets[batch_id]); i++) {
    output_mask[batch_id*mask_stride + i] = 1;
  }
  for (int i = 0; i < (offsets[batch_id + 1] - offsets[batch_id]) * inner_size; i++) {
    output[batch_id * output_stride + i] = input[offsets[batch_id] * inner_size + i];
  }
}

template<typename T>
void add_padding_mask_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    int* output_mask, // [batch_size x max(input.nested_size(1))]
    const int* offsets, // [batch_size]
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;

  add_padding_mask<float><<<grid, 1, 0, stream>>>(
      input,
      output,
      output_mask,
      offsets,
      batch_size,
      mask_stride,
      output_stride,
      inner_size);
}

template void add_padding_mask_kernelLauncher<float>(
    float* input,
    float* output,
    int* output_mask,
    const int* offsets,
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream);

template<typename T>
__global__
void remove_padding(
    const T* input,
    T* output,
    const int* offsets,
    const int batch_size,
    const int output_stride,
    const int inner_size)
{
  const int batch_id  = blockIdx.x;
  const int grain_size = blockDim.x;
  const int tid = threadIdx.x;
  const int range = (offsets[batch_id + 1] - offsets[batch_id]) * inner_size;
  const int num_chunks = range / grain_size;
  for (int id = 0; id < num_chunks; id++) {
    output[offsets[batch_id] * inner_size + id * grain_size + tid]
     = input[batch_id * output_stride + id * grain_size + tid];
  }
  const int leftover = num_chunks * grain_size;
  if (leftover + tid < range) {
    output[offsets[batch_id] * inner_size + leftover + tid]
     = input[batch_id * output_stride + leftover + tid];
  }
}

template<typename T>
void remove_padding_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    const int* offsets, // [batch_size]
    const int batch_size,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;

  remove_padding<float><<<grid, 1024, 0, stream>>>(
      input,
      output,
      offsets,
      batch_size,
      output_stride,
      inner_size);
}

template void remove_padding_kernelLauncher<float>(
    float* input,
    float* output,
    const int* offsets,
    const int batch_size,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream);
}
}
