#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <nestedtensor/csrc/cuda/padding.h>
#include <stdio.h>

namespace nested_tensor {
namespace cuda {

template<typename T>
__global__
void add_padding_1(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    const int* input_strides,
    int input_dim,
    const int* output_sizes,
    const int* output_strides,
    const int batch_size)
{
  const int batch_id  = blockIdx.x;
  const int grid_id  = blockIdx.y;
  const int tid = threadIdx.x + grid_id * 256;
  const int grainsize = 16 * 256;
  const int batch_input_offset = offsets[batch_id];
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int numel_i = sizes_i[0];
  const int batch_output_offset = batch_id * output_sizes[1];
  for (int ii = 0; ii < (output_sizes[1] / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int output_offset = batch_output_offset + i;
    if (i < sizes_i[0]) {
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
  const int i = (output_sizes[1] / grainsize) * grainsize + tid;
  if (i < output_sizes[1]) {
    const int output_offset = batch_output_offset + i;
    if (i < sizes_i[0]) {
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
}

template<typename T>
__global__
void add_padding_2(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    const int* input_strides,
    int input_dim,
    const int* output_sizes,
    const int* output_strides,
    const int batch_size)
{
  const int batch_id  = blockIdx.x;
  const int grid_id  = blockIdx.y;
  const int tid = threadIdx.x + grid_id * 256;
  const int grainsize = 16 * 256;
  const int offset = offsets[batch_id];
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int numel_i = sizes_i[0] * sizes_i[1];
  const int output_offset = batch_id * output_sizes[1] * output_sizes[2];
  const int output_numel = output_sizes[1] * output_sizes[2];
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes[2]);
    const int i1 = i % output_sizes[2];
    if (i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes[2]);
    const int i1 = i % output_sizes[2];
    if (i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template<typename T>
__global__
void add_padding_3(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    const int* input_strides,
    int input_dim,
    const int* output_sizes,
    const int* output_strides,
    const int batch_size)
{
  const int batch_id  = blockIdx.x;
  const int grid_id  = blockIdx.y;
  const int tid = threadIdx.x + grid_id * 256;
  const int grainsize = 16 * 256;
  const int offset = offsets[batch_id];
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int* strides_i = input_strides + batch_id * input_dim;
  const int numel_i = sizes_i[0] * sizes_i[1] * sizes_i[2];
  const int output_numel = output_sizes[1] * output_sizes[2] * output_sizes[3];
  if (threadIdx.x == 0 && grid_id == 0) {
  printf("output_sizes: (%d, %d, %d, %d) output_strides: (%d, %d, %d, %d)\n",
      output_sizes[0],
      output_sizes[1],
      output_sizes[2],
      output_sizes[3],
      output_strides[0],
      output_strides[1],
      output_strides[2],
      output_strides[3]);
  }
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes[2] * output_sizes[3]);
    const int i1 = (i % (output_sizes[2] * output_sizes[3])) / output_sizes[3];
    const int i2 = i % output_sizes[3];
    const int output_offset = batch_id * output_strides[0] + 
                                    i0 * output_strides[1] +
                                    i1 * output_strides[2] +
                                    i2 * output_strides[3];
    if (i0 < sizes_i[0] && i1 < sizes_i[1] && i2 < sizes_i[2]) {
      const int input_offset = offset +
                               i0 * strides_i[0] +
                               i1 * strides_i[1] +
                               i2 * strides_i[2];
  printf("%d -  i, index: (%d, %d, %d, %d): %d - %d\n",
      i,
      batch_id,
      i0,
      i1,
      i2,
      output_offset,
      input_offset);
      output[output_offset] = input[input_offset];
    } else {
      output[output_offset] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes[2] * output_sizes[3]);
    const int i1 = (i % (output_sizes[2] * output_sizes[3])) / output_sizes[3];
    const int i2 = i % output_sizes[3];
//  if (threadIdx.x == 0 && grid_id == 0) {
//  }
    const int output_offset = batch_id * output_strides[0] + 
                                    i0 * output_strides[1] +
                                    i1 * output_strides[2] +
                                    i2 * output_strides[3];
    if (i0 < sizes_i[0] && i1 < sizes_i[1] && i2 < sizes_i[2]) {
      const int input_offset = offset +
                               i0 * strides_i[0] +
                               i1 * strides_i[1] +
                               i2 * strides_i[2];
  printf("%d -  i, index: (%d, %d, %d, %d): %d - %d\n",
      i,
      batch_id,
      i0,
      i1,
      i2,
      output_offset,
      input_offset);
      output[output_offset] = input[input_offset];
    } else {
      output[output_offset] = padding_value;
    }
  }
}

template<typename T>
void add_padding_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    const int* input_strides,
    int input_dim,
    const int* output_sizes,
    const int* output_strides,
    const int batch_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;
  grid.y = 16;
  if (input_dim == 1) {
    add_padding_1<T><<<grid, 256, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_strides,
        input_dim,
        output_sizes,
        output_strides,
        batch_size);
  }
  if (input_dim == 2) {
    add_padding_2<T><<<grid, 256, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_strides,
        input_dim,
        output_sizes,
        output_strides,
        batch_size);
  }
  if (input_dim == 3) {
    add_padding_3<T><<<grid, 256, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_strides,
        input_dim,
        output_sizes,
        output_strides,
        batch_size);
  }
}

template void add_padding_kernelLauncher<float>(
    float* input,
    float* output,
    float padding_value,
    const int* offsets,
    const int* input_sizes,
    const int* input_strides,
    int input_dim,
    const int* output_sizes,
    const int* output_strides,
    const int batch_size,
    const hipStream_t stream);

template void add_padding_kernelLauncher<c10::Half>(
    c10::Half* input,
    c10::Half* output,
    c10::Half padding_value,
    const int* offsets,
    const int* input_sizes,
    const int* input_strides,
    int input_dim,
    const int* output_sizes,
    const int* output_strides,
    const int batch_size,
    const hipStream_t stream);

template<typename T>
__global__
void add_padding_mask(
    const T* input,
    T* output,
    int* output_mask,
    const int* offsets,
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size)
{
  const int batch_id  = blockIdx.x;
  for (int i = 0; i < (offsets[batch_id + 1] - offsets[batch_id]); i++) {
    output_mask[batch_id*mask_stride + i] = 1;
  }
  for (int i = 0; i < (offsets[batch_id + 1] - offsets[batch_id]) * inner_size; i++) {
    output[batch_id * output_stride + i] = input[offsets[batch_id] * inner_size + i];
  }
}

template<typename T>
void add_padding_mask_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    int* output_mask, // [batch_size x max(input.nested_size(1))]
    const int* offsets, // [batch_size]
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;

  add_padding_mask<float><<<grid, 1, 0, stream>>>(
      input,
      output,
      output_mask,
      offsets,
      batch_size,
      mask_stride,
      output_stride,
      inner_size);
}

template void add_padding_mask_kernelLauncher<float>(
    float* input,
    float* output,
    int* output_mask,
    const int* offsets,
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream);

template<typename T>
__global__
void remove_padding(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size)
{
  const int batch_id  = blockIdx.x;
  const int grid_id  = blockIdx.y;
  const int tid = threadIdx.x + grid_id * 256;
  const int grainsize = 16 * 256;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1] * sizes_i[2];
  int input_offset = batch_id * input_sizes[1] * input_sizes[2] * input_sizes[3];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (sizes_i[1] * sizes_i[2]);
    const int i1 = (i % (sizes_i[1] * sizes_i[2])) / sizes_i[2];
    const int i2 = i % sizes_i[2];
    const int i0_offset = i0 * input_sizes[2] * input_sizes[3];
    const int i1_offset = i1 * input_sizes[3];
    output[offset + i] = input[input_offset + i0_offset + i1_offset + i2];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i0 = i / (sizes_i[1] * sizes_i[2]);
    const int i1 = (i % (sizes_i[1] * sizes_i[2])) / sizes_i[2];
    const int i2 = i % sizes_i[2];
    const int i0_offset = i0 * input_sizes[2] * input_sizes[3];
    const int i1_offset = i1 * input_sizes[3];
    output[offset + i] = input[input_offset + i0_offset + i1_offset + i2];
  }
}

template<typename T>
void remove_padding_kernelLauncher(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;
  grid.y = 16;

  remove_padding<T><<<grid, 256, 0, stream>>>(
    input,
    output,
    offsets,
    input_sizes,
    output_sizes,
    output_dim,
    batch_size);
}

template void remove_padding_kernelLauncher<float>(
    const float* input,
    float* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size,
    const hipStream_t stream);

template void remove_padding_kernelLauncher<c10::Half>(
    const c10::Half* input,
    c10::Half* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size,
    const hipStream_t stream);
}
}
