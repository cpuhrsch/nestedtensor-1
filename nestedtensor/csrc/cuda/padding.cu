#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <nestedtensor/csrc/cuda/padding.h>
#include <stdio.h>

namespace nested_tensor {
namespace cuda {

template<typename T>
__global__
void add_padding_1(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const int* output_sizes,
    const int batch_size)
{
  const int batch_id  = blockIdx.x;
  const int grid_id  = blockIdx.y;
  const int tid = threadIdx.x + grid_id * 256;
  const int grainsize = 16 * 256;
  const int batch_input_offset = offsets[batch_id];
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int numel_i = sizes_i[0];
  const int batch_output_offset = batch_id * output_sizes[1];
  for (int ii = 0; ii < (output_sizes[1] / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int output_offset = batch_output_offset + i;
    if (i < sizes_i[0]) {
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
  const int i = (output_sizes[1] / grainsize) * grainsize + tid;
  if (i < output_sizes[1]) {
    const int output_offset = batch_output_offset + i;
    if (i < sizes_i[0]) {
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
}

template<typename T>
__global__
void add_padding_2(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const int* output_sizes,
    const int batch_size)
{
  const int batch_id  = blockIdx.x;
  const int grid_id  = blockIdx.y;
  const int tid = threadIdx.x + grid_id * 256;
  const int grainsize = 16 * 256;
  const int offset = offsets[batch_id];
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int numel_i = sizes_i[0] * sizes_i[1];
  const int output_offset = batch_id * output_sizes[1] * output_sizes[2];
  const int output_numel = output_sizes[1] * output_sizes[2];
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes[2]);
    const int i1 = i % output_sizes[2];
    if (i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes[2]);
    const int i1 = i % output_sizes[2];
    if (i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template<typename T>
__global__
void add_padding_3(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const int* output_sizes,
    const int batch_size)
{
  const int batch_id  = blockIdx.x;
  const int grid_id  = blockIdx.y;
  const int tid = threadIdx.x + grid_id * 256;
  const int grainsize = 16 * 256;
  const int offset = offsets[batch_id];
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int numel_i = sizes_i[0] * sizes_i[1] * sizes_i[2];
  const int output_offset = batch_id * output_sizes[1] * output_sizes[2] * output_sizes[3];
  const int output_numel = output_sizes[1] * output_sizes[2] * output_sizes[3];
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes[2] * output_sizes[3]);
    const int i1 = (i % (output_sizes[2] * output_sizes[3])) / output_sizes[3];
    const int i2 = i % output_sizes[3];
    if (i0 < sizes_i[0] && i1 < sizes_i[1] && i2 < sizes_i[2]) {
      const int input_offset = offset + i0 * (sizes_i[1] * sizes_i[2]) + i1 * sizes_i[2] + i2;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes[2] * output_sizes[3]);
    const int i1 = (i % (output_sizes[2] * output_sizes[3])) / output_sizes[3];
    const int i2 = i % output_sizes[3];
    if (i0 < sizes_i[0] && i1 < sizes_i[1] && i2 < sizes_i[2]) {
      const int input_offset = offset + i0 * (sizes_i[1] * sizes_i[2]) + i1 * sizes_i[2] + i2;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template<typename T>
void add_padding_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const int* output_sizes,
    const int batch_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;
  grid.y = 16;
  if (input_dim == 1) {
    add_padding_1<T><<<grid, 256, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes,
        batch_size);
  }
  if (input_dim == 2) {
    add_padding_2<T><<<grid, 256, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes,
        batch_size);
  }
  if (input_dim == 3) {
    add_padding_3<T><<<grid, 256, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes,
        batch_size);
  }
}

template void add_padding_kernelLauncher<float>(
    float* input,
    float* output,
    float padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const int* output_sizes,
    const int batch_size,
    const hipStream_t stream);

template void add_padding_kernelLauncher<c10::Half>(
    c10::Half* input,
    c10::Half* output,
    c10::Half padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const int* output_sizes,
    const int batch_size,
    const hipStream_t stream);

template<typename T>
__global__
void add_padding_mask(
    const T* input,
    T* output,
    int* output_mask,
    const int* offsets,
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size)
{
  const int batch_id  = blockIdx.x;
  for (int i = 0; i < (offsets[batch_id + 1] - offsets[batch_id]); i++) {
    output_mask[batch_id*mask_stride + i] = 1;
  }
  for (int i = 0; i < (offsets[batch_id + 1] - offsets[batch_id]) * inner_size; i++) {
    output[batch_id * output_stride + i] = input[offsets[batch_id] * inner_size + i];
  }
}

template<typename T>
void add_padding_mask_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    int* output_mask, // [batch_size x max(input.nested_size(1))]
    const int* offsets, // [batch_size]
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size;

  add_padding_mask<float><<<grid, 1, 0, stream>>>(
      input,
      output,
      output_mask,
      offsets,
      batch_size,
      mask_stride,
      output_stride,
      inner_size);
}

template void add_padding_mask_kernelLauncher<float>(
    float* input,
    float* output,
    int* output_mask,
    const int* offsets,
    const int batch_size,
    const int mask_stride,
    const int output_stride,
    const int inner_size,
    const hipStream_t stream);

template<typename T, int num_threads, int grid_y>
__global__
void remove_padding(
    const T* input,
    T* output,
    const int* offsets,
    const int* output_sizes,
    int output_dim,
    const int batch_size,
    const int input_sizes_1_2_3,
    const int input_sizes_2_3,
    const int input_sizes_3)
{
  const int batch_id  = blockIdx.x / grid_y;
  const int grid_id  = blockIdx.x % grid_y;
  const int tid = threadIdx.x + grid_id * num_threads;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int size_0 = sizes_i[0];
  const int size_1 = sizes_i[1];
  const int size_2 = sizes_i[2];
  const int size_1_2 = size_1 * size_2;
  const int numel_i = size_0 * size_1_2;
  int input_offset = batch_id * input_sizes_1_2_3;
  int num_chunks = input_sizes_1_2_3 / (grid_y * num_threads);
  for (int ii = 0; ii < num_chunks; ii++) {
    const int i = ii * grid_y * num_threads + tid;
    const int i0 = i / (input_sizes_2_3);
    const int i1 = (i % (input_sizes_2_3)) / input_sizes_3;
    const int i2 = i % input_sizes_3;
    if (i0 < size_0 && i1 < size_1 && i2 < size_2) {
      output[offset + i0 * size_1_2 + i1 * size_2 + i2] = input[input_offset + i];
    }
  }
  const int i = (input_sizes_1_2_3 / (grid_y * num_threads)) * (grid_y * num_threads) + tid;
  if (i < input_sizes_1_2_3) {
    const int i0 = i / (input_sizes_2_3);
    const int i1 = (i % (input_sizes_2_3)) / input_sizes_3;
    const int i2 = i % input_sizes_3;
    if (i0 < size_0 && i1 < size_1 && i2 < size_2) {
      output[offset + i0 * size_1_2 + i1 * size_2 + i2] = input[input_offset + i];
    }
  }
}

template<typename T>
void remove_padding_kernelLauncher(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = batch_size * 16;
  const int input_sizes_1_2_3 = input_sizes[1] * input_sizes[2] * input_sizes[3];
  const int input_sizes_2_3 = input_sizes[2] * input_sizes[3];
  const int input_sizes_3 = input_sizes[3];

  remove_padding<T, 256, 16><<<grid, 256, 0, stream>>>(
    input,
    output,
    offsets,
    output_sizes,
    output_dim,
    batch_size,
    input_sizes_1_2_3,
    input_sizes_2_3,
    input_sizes_3);
}

template void remove_padding_kernelLauncher<float>(
    const float* input,
    float* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size,
    const hipStream_t stream);

template void remove_padding_kernelLauncher<c10::Half>(
    const c10::Half* input,
    c10::Half* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size,
    const hipStream_t stream);
}
}
