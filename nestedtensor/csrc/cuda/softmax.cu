#include "hip/hip_runtime.h"
/*
* Copyright (c) 2020-2021, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.

Changes in comparison to original.
- Removed any code unrelated to softmax

*/

#include <nestedtensor/csrc/cuda/softmax.h>
#include <stdio.h>
#include <c10/util/Half.h>

namespace fastertransformer 
{

#define FINAL_MASK 0xffffffff

template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
    #pragma unroll
    for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
    return val;
}

/* Calculate the sum of all elements in a block */
template <typename T>
    __inline__ __device__
T blockReduceSum(T val)
{
    static __shared__ T shared[32]; 
    int lane = threadIdx.x & 0x1f; 
    int wid = threadIdx.x >> 5;  

    val = warpReduceSum<T>(val);

    if(lane == 0)
    shared[wid] = val;

    __syncthreads();

    val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)(0.0f);
    val = warpReduceSum<T>(val);
                                
    return val;
}

template <typename T>
    __inline__ __device__
T warpReduceMax(T val)
{
    #pragma unroll
    for(int mask = 16; mask > 0; mask >>= 1)
    val = max(val, __shfl_xor_sync(FINAL_MASK, val, mask, 32));
    return val;
}

/* Calculate the maximum of all elements in a block */
template <typename T>
    __inline__ __device__
T blockReduceMax(T val)
{
    static __shared__ T shared[32]; 
    int lane = threadIdx.x & 0x1f; // in-warp idx
    int wid = threadIdx.x >> 5;  // warp idx

    val = warpReduceMax(val); // get maxx in each warp

    if(lane == 0) // record in-warp maxx by warp Idx
    shared[wid] = val;

    __syncthreads();


    val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : -1e20f;
    val = warpReduceMax(val);

    return val;
}

__inline__ __device__
int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4)
{
  return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

template <typename T>
__global__
void softmax_kernel(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, 
  const T scalar)
{
    int batch_id = blockIdx.x / head_num;
    int qk_offset = blockIdx.x * seq_len * seq_len;
    int mask_offset = batch_id * seq_len * seq_len;

    __shared__ float s_sum, s_max;

    for(int i = 0; i < seq_len; ++i)
    {
      float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
      float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
      mask_val = (1.0f - mask_val) * -10000.0f;

      float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + mask_val): -1e20f;

      float max_val = blockReduceMax<float>(tmp);

      if(threadIdx.x == 0)
        s_max = max_val;
      __syncthreads();

      qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

      float sum_val = blockReduceSum<float>(qk);

      if(threadIdx.x == 0)
      {
        s_sum = sum_val + 1e-6f;
      }
      __syncthreads();

      if(threadIdx.x < seq_len)
        qk_buf_[threadIdx.x + qk_offset] = (T)(qk / s_sum);

      qk_offset += seq_len;
      mask_offset += seq_len;
    }
}


template <typename T>
__global__
void softmax_kernel_v2(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, 
  const int seq_len, const float scalar)
{
    int batch_id = blockIdx.x / head_num / seq_len;
    int seq_id = blockIdx.x % seq_len;
    int qk_offset = blockIdx.x * seq_len;
    int mask_offset = batch_id * seq_len * seq_len + seq_id * seq_len;

    __shared__ float s_sum, s_max;

    float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
    float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
    mask_val = (1.0f - mask_val) * -10000.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + mask_val) : -1e20f;
    float max_val = blockReduceMax<float>(tmp);
    if(threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    float qk_tmp = threadIdx.x < seq_len ? __expf((float)(tmp - s_max)) : 0.0f;
    float sum_val = blockReduceSum<float>(qk_tmp);

    if(threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if(threadIdx.x < seq_len)
      qk_buf_[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

//grid = (seq_len/word_per_thread, batch_size, head_num)
//block.x = max(32, (seq_len + 31)/32*32)
template <typename T>
__global__
void softmax_kernel_v3(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, const T scalar)
{
    
  bool qual = threadIdx.x < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    float tmp = -1e20f;
    int qk_offset;
    __shared__ float s_mean, s_max;
    if (qual){
      qk_offset = ((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len + threadIdx.x;
      int mask_offset = (blockIdx.y * seq_len + seq_id) * seq_len + threadIdx.x;

      float qk = static_cast<float>(qk_buf_[qk_offset]);
      float mask_val = static_cast<float>(__ldg(&attr_mask[mask_offset]));

      mask_val = (1.0f - mask_val) * -10000.0f;

      tmp = qk * static_cast<float>(scalar) + mask_val;
    }

    float max_val = blockReduceMax<float>(tmp);
    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();
    
    float qk_tmp = qual ? __expf(tmp - s_max) : 0.0f;
    float sum_val = blockReduceSum<float>(qk_tmp);
    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();
    
    if(qual)
      qk_buf_[qk_offset] = (T)(qk_tmp * s_mean);
  }
}  


//grid = (seq_len/word_per_thread, batch_size, head_num)
//block.x = max(32, (seq_len/2 + 31)/32*32)
//seq_len % 2 == 0
template <>
__global__
void softmax_kernel_v3(half* qk_buf_, const half* attr_mask, 
                      const int batch_size, const int head_num, 
                      const int seq_len, const half scalar)
{
  int threadIdx2 = threadIdx.x << 1;
  bool qual = threadIdx2 < seq_len;
  half2* qk_buf_half2Ptr = (half2*) qk_buf_;
  const half2* attr_mask_half2Ptr = (const half2*) attr_mask;
  __shared__ float s_mean, s_max;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int qk_offset;
    half2 tmp = __float2half2_rn(0.0f);

    float max_val = -1e20f;
    half2 qk;
    if (qual){ 
      qk_offset = ((((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len) >> 1) + threadIdx.x;
      int mask_offset = (((blockIdx.y * seq_len + seq_id) * seq_len) >> 1) + threadIdx.x;

      qk = qk_buf_half2Ptr[qk_offset];
      half2 mask_val = __ldg(&attr_mask_half2Ptr[mask_offset]);
      half2 mask_val_tmp = __hmul2(__hsub2(__float2half2_rn(1.0f), mask_val), __float2half2_rn(-10000.0f));
      tmp = __hadd2(__hmul2(__half2half2(scalar), qk), mask_val_tmp);
      max_val = fmax((float)tmp.x, (float)tmp.y);
    }
    
    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();
    
    if (qual){
      tmp = h2exp(__hsub2(tmp, __float2half2_rn(s_max)));
    }
    float sum_val = blockDim.x <= 32 ? warpReduceSum((float)(tmp.x + tmp.y)) : blockReduceSum<float>((float)(tmp.x + tmp.y));

    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();

    if(qual){
      qk = __hmul2(tmp, __float2half2_rn(s_mean));
      qk_buf_half2Ptr[qk_offset] = qk;
    }
  }
}

template <typename T>
__global__
void softmax_kernel_v3_LE32(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, const T scalar)
{
  bool qual = threadIdx.x < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int qk_offset;
    __shared__ float s_mean, s_max;
    float tmp = -1e20f;
    if (qual){
      qk_offset = ((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len + threadIdx.x;
      int mask_offset = (blockIdx.y * seq_len + seq_id) * seq_len + threadIdx.x;

      float qk = static_cast<float>(qk_buf_[qk_offset]);
      float mask_val = static_cast<float>(__ldg(&attr_mask[mask_offset]));

      mask_val = (1.0f - mask_val) * -10000.0f;

      tmp = static_cast<float>(qk) * static_cast<float>(scalar) + mask_val;
    }
    float max_val = warpReduceMax<float>(tmp);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    tmp = qual ? __expf(tmp - s_max) : 0.0f;
    float sum_val = warpReduceSum<float>(tmp);

    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();

    if(qual)
      qk_buf_[qk_offset] = (T)(tmp * s_mean);
  }
}

template<typename T>
void attn_softmax_kernelLauncher(
  T* buffer,
  const T* attr_mask,
  const int batch_size,
  const int seq_len,
  const int head_num,
  const T scalar,
  hipStream_t stream)
{
  dim3 grid, block;
  //deal with odd seq_len
  if (seq_len % 2 != 0){
    if(seq_len <= 32)
      block.x = 32;
    else if(seq_len > 32 && seq_len <= 64)
      block.x = 64;
    else if(seq_len > 64 && seq_len <= 128)
      block.x = 128;
    else if(seq_len > 128 && seq_len <= 256)
      block.x = 256;
    else if(seq_len > 256 && seq_len <= 512)
      block.x = 512;
    else
      block.x = 1024;

    if(batch_size * head_num <= 120)
    {
      grid.x = batch_size * head_num * seq_len;
      softmax_kernel_v2<T><<<grid, block, 0, stream>>>(buffer, attr_mask, batch_size, head_num, seq_len, scalar);
    }
    else
    {
      grid.x = batch_size * head_num;
      softmax_kernel<T><<<grid, block, 0, stream>>>(buffer, attr_mask, batch_size, head_num, seq_len, scalar);
    }
  }
  //deal with even seq_len 
  else{
    grid.x = seq_len;
    if (batch_size * head_num > 360)
      grid.x = ceil(float(seq_len)/32.0f);
    grid.y = batch_size;
    grid.z = head_num;
    if (seq_len <= 32){
      block.x = 32;
      softmax_kernel_v3_LE32<T><<<grid, block, 0, stream>>>(buffer, attr_mask, batch_size, head_num, seq_len, scalar);
    }
    else{
      if (sizeof(T) == 2){
        block.x = (seq_len/2 + 31)/32*32;
        softmax_kernel_v3<<<grid, block, 0, stream>>>(buffer, attr_mask, batch_size, head_num, seq_len, scalar);
      }
      else{
        block.x = (seq_len + 31)/32*32;
        softmax_kernel_v3<T><<<grid, block, 0, stream>>>(buffer, attr_mask, batch_size, head_num, seq_len, scalar);
      }
    }
    grid.x = grid.y = grid.z = 1;
  }
}

template void attn_softmax_kernelLauncher(
    float* buffer,
    const float* attr_mask,
    const int batch_size,
    const int seq_len,
    const int head_num,
    const float scalar,
    hipStream_t stream);
    
template void attn_softmax_kernelLauncher(
    half* buffer,
    const half* attr_mask,
    const int batch_size,
    const int seq_len,
    const int head_num,
    const half scalar,
    hipStream_t stream);

template void attn_softmax_kernelLauncher(
    c10::Half* buffer,
    const c10::Half* attr_mask,
    const int batch_size,
    const int seq_len,
    const int head_num,
    const c10::Half scalar,
    hipStream_t stream);
      
} // namespace fastertransformer
