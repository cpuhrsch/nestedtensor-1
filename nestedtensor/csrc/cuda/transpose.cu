#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <nestedtensor/csrc/cuda/transpose.h>
#include <stdio.h>

namespace nested_tensor {
namespace cuda {

template<typename T, int num_threads_sqrt>
__global__
void transpose_nchw_nhwc(
    T* input,
    T* output,
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int num_channel,
    const int* image_numel)
{
  __shared__ T tile[num_threads_sqrt][num_threads_sqrt + 1];
  const int block_id  = blockIdx.x;
  const int tid2 = threadIdx.x / 32;
  const int tid3 = threadIdx.x % 32;
  int batch_id = threadIdx.x % 32;
  bool found = false;
  while (batch_id < batch_size) {
    if (block_offsets[batch_id] <= block_id && 
        block_id < block_offsets[batch_id + 1]) {
      found = true;
      break;
    }
    batch_id += 32;
  }
  if (!found) {
    batch_id = 0;
  }
  // TODO: Parameterize on warp size instead of assuming 32.
  for (int warp_offset = 16; warp_offset > 0; warp_offset /= 2)
      batch_id = batch_id | __shfl_down_sync(0xFFFFFFFF, batch_id, warp_offset);
  batch_id = __shfl_sync(0xFFFFFFFF, batch_id, 0, 32);

  const int grain_size = num_threads_sqrt;
  const int size2 = num_channel;
  const int size3 = image_numel[batch_id];
  const int block_offset = block_offsets[batch_id];
  const int offset = offsets[batch_id];

  const int num_chunks_3 = (size3  + grain_size - 1) / grain_size;
  const int current_block = block_id - block_offset;
  const int current_block_mod = (current_block % num_chunks_3) * grain_size;
  const int current_block_div = (current_block / num_chunks_3) * grain_size;
  const int offset1_tid2 = (current_block_mod) + tid2;
  const int offset2_tid2 = (current_block_div) + tid2;
  const int offset1_tid3 = (current_block_mod) + tid3;
  const int offset2_tid3 = (current_block_div) + tid3;
  const int ii3 = offset1_tid3;
#pragma unroll
  for (int sub = 0; sub < 4; sub++) {
    const int ii2 = offset2_tid2 + sub * 8;
    if (ii2 < size2 && ii3 < size3) {
      const int ii = ii2 * size3 + ii3;
      tile[tid2 + sub * 8][tid3] = input[offset + ii];
    }
  }

  __syncthreads();

  const int ii21 = offset2_tid3;
#pragma unroll
  for (int sub = 0; sub < 4; sub++) {
    const int ii31 = offset1_tid2 + sub * 8;
    if (ii21 < size2 && ii31 < size3) {
      const int ii1 = ii21 * size3 + ii31;
      const int j = (ii1 % size3) * size2;
      const int i = (ii1 / size3);
      output[offset + j + i] = tile[tid3][tid2 + sub * 8];
    }
  }
}

template <typename T>
void transpose_nchw_nhwc_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int block_numel,
    const int num_channel,
    const int* image_numel,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = block_numel;

  transpose_nchw_nhwc<T, 32><<<grid, 256, 0, stream>>>(
      input,
      output,
      block_offsets,
      offsets,
      batch_size,
      num_channel,
      image_numel);
}

template void transpose_nchw_nhwc_kernelLauncher<c10::Half>(
    c10::Half* input,
    c10::Half* output,
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int block_numel,
    const int num_channel,
    const int* image_numel,
    const hipStream_t stream);

template void transpose_nchw_nhwc_kernelLauncher<float>(
    float* input,
    float* output,
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int block_numel,
    const int num_channel,
    const int* image_numel,
    const hipStream_t stream);

template<typename T, int num_threads_sqrt>
__global__
void transpose_nhwc_nchw(
    T* input,
    T* output,
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int num_channel,
    const int* image_numel)
{
  __shared__ T tile[num_threads_sqrt][num_threads_sqrt + 1];
  const int block_id  = blockIdx.x;
  const int tid2 = threadIdx.x / 32;
  const int tid3 = threadIdx.x % 32;
  int batch_id = threadIdx.x % 32;
  bool found = false;
  while (batch_id < batch_size) {
    if (block_offsets[batch_id] <= block_id && 
        block_id < block_offsets[batch_id + 1]) {
      found = true;
      break;
    }
    batch_id += 32;
  }
  if (!found) {
    batch_id = 0;
  }
  // TODO: Parameterize on warp size instead of assuming 32.
  for (int warp_offset = 16; warp_offset > 0; warp_offset /= 2)
      batch_id = batch_id | __shfl_down_sync(0xFFFFFFFF, batch_id, warp_offset);
  batch_id = __shfl_sync(0xFFFFFFFF, batch_id, 0, 32);

  const int grain_size = num_threads_sqrt;
  const int size2 = image_numel[batch_id];
  const int size3 = num_channel;
  const int block_offset = block_offsets[batch_id];
  const int offset = offsets[batch_id];

  const int num_chunks_3 = (size3  + grain_size - 1) / grain_size;
  const int current_block = block_id - block_offset;
  const int current_block_mod = (current_block % num_chunks_3) * grain_size;
  const int current_block_div = (current_block / num_chunks_3) * grain_size;
  const int offset1_tid2 = (current_block_mod) + tid2;
  const int offset2_tid2 = (current_block_div) + tid2;
  const int offset1_tid3 = (current_block_mod) + tid3;
  const int offset2_tid3 = (current_block_div) + tid3;
  const int ii3 = offset1_tid3;
  if (ii3 < size3) {
#pragma unroll
    for (int sub = 0; sub < 4; sub++) {
      const int ii2 = offset2_tid2 + sub * 8;
      if (ii2 < size2) {
        const int ii = ii2 * size3 + ii3;
        tile[tid2 + sub * 8][tid3] = input[offset + ii];
      }
    }
  }

  __syncthreads();

  const int ii21 = offset2_tid3;
  if (ii21 < size2) {
#pragma unroll
    for (int sub = 0; sub < 4; sub++) {
      const int ii31 = offset1_tid2 + sub * 8;
      if (ii31 < size3) {
        const int ii1 = ii21 * size3 + ii31;
        const int j = (ii1 % size3) * size2;
        const int i = (ii1 / size3);
        output[offset + j + i] = tile[tid3][tid2 + sub * 8];
      }
    }
  }
}

template <typename T>
void transpose_nhwc_nchw_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int block_numel,
    const int num_channel,
    const int* image_numel,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = block_numel;

  transpose_nhwc_nchw<T, 32><<<grid, 256, 0, stream>>>(
      input,
      output,
      block_offsets,
      offsets,
      batch_size,
      num_channel,
      image_numel);
}

template void transpose_nhwc_nchw_kernelLauncher<c10::Half>(
    c10::Half* input,
    c10::Half* output,
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int block_numel,
    const int num_channel,
    const int* image_numel,
    const hipStream_t stream);

template void transpose_nhwc_nchw_kernelLauncher<float>(
    float* input,
    float* output,
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int block_numel,
    const int num_channel,
    const int* image_numel,
    const hipStream_t stream);

}
} // namespace nested_tensor
