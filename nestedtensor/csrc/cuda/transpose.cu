#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <nestedtensor/csrc/cuda/transpose.h>
#include <stdio.h>

namespace nested_tensor {
namespace cuda {

template<int num_threads_sqrt>
__global__
void transpose(
    c10::Half* input,
    c10::Half* output,
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int* size_dim2,
    const int* size_dim3)
{
  __shared__ c10::Half tile[num_threads_sqrt][num_threads_sqrt + 1];
  __shared__ int batch_id;
  const int block_id  = blockIdx.x;
  const int tid2 = threadIdx.x;
  const int tid3 = threadIdx.y;
  int batch_id_search = tid2 * 32 + tid3;
  while (batch_id_search < batch_size) {
    if (block_offsets[batch_id_search] <= block_id && 
        block_id < block_offsets[batch_id_search + 1]) {
      batch_id = batch_id_search;
    }
    batch_id_search += 256;
  }
  __syncthreads();
  const int grain_size = num_threads_sqrt;
  const int size2 = size_dim2[batch_id];
  const int size3 = size_dim3[batch_id];
  const int block_offset = block_offsets[batch_id];
  const int offset = offsets[batch_id];

  const int num_chunks_3 = (size3  + grain_size - 1) / grain_size;
  const int current_block = block_id - block_offset;
  const int ii3 = (current_block % num_chunks_3) * grain_size + tid3;
  for (int sub = 0; sub < 4; sub++) {
    const int ii2 = (current_block / num_chunks_3) * grain_size + tid2 + sub * 8;
    if (ii2 < size2 && ii3 < size3) {
      const int ii = ii2 * size3 + ii3;
      tile[tid2 + sub * 8][tid3] = __ldg(reinterpret_cast<const __half*>(input) + offset + ii);
    }
  }

  __syncthreads();

  const int ii21 = (current_block / num_chunks_3) * grain_size + tid3;
  for (int sub = 0; sub < 4; sub++) {
    const int ii31 = (current_block % num_chunks_3) * grain_size + tid2 + sub * 8;
    if (ii21 < size2 && ii31 < size3) {
      const int ii1 = ii21 * size3 + ii31;
      const int j = (ii1 % size3) * size2;
      const int i = (ii1 / size3);
      output[offset + j + i] = tile[tid3][tid2 + sub * 8];
    }
  }
}

void transpose_kernelLauncher(
    c10::Half* input, // [batch_size x None]
    c10::Half* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int block_numel,
    const int* size_dim2,
    const int* size_dim3,
    const hipStream_t stream)
{
  dim3 grid;
  // Actually is batch size.
  grid.x = block_numel,

  transpose<32><<<grid, dim3(8, 32), 0, stream>>>(
      input,
      output,
      block_offsets,
      offsets,
      batch_size,
      size_dim2,
      size_dim3);
}

}
} // namespace nested_tensor
