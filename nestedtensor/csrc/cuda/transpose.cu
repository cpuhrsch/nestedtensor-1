#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <nestedtensor/csrc/cuda/transpose.h>
#include <stdio.h>

namespace nested_tensor {
namespace cuda {

template<int num_threads_sqrt>
__global__
void transpose(
    c10::Half* input,
    c10::Half* output,
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int* size_dim2,
    const int* size_dim3)
{
  __shared__ c10::Half tile[num_threads_sqrt][num_threads_sqrt + 1];
  const int block_id  = blockIdx.x;
  const int tid2 = threadIdx.x / 32;
  const int tid3 = threadIdx.x % 32;
  int batch_id = threadIdx.x % 32;
  bool found = false;
  while (batch_id < batch_size) {
    if (block_offsets[batch_id] <= block_id && 
        block_id < block_offsets[batch_id + 1]) {
      found = true;
      break;
    }
    batch_id += 32;
  }
  if (!found) {
    batch_id = 0;
  }
  // TODO: Parameterize on warp size instead of assuming 32.
  for (int warp_offset = 16; warp_offset > 0; warp_offset /= 2)
      batch_id = batch_id | __shfl_down_sync(0xFFFFFFFF, batch_id, warp_offset);
  batch_id = __shfl_sync(0xFFFFFFFF, batch_id, 0, 32);

  const int grain_size = num_threads_sqrt;
  const int size2 = size_dim2[batch_id];
  const int size3 = size_dim3[batch_id];
  const int block_offset = block_offsets[batch_id];
  const int offset = offsets[batch_id];

  const int num_chunks_3 = (size3  + grain_size - 1) / grain_size;
  const int current_block = block_id - block_offset;
  const int current_block_mod = (current_block % num_chunks_3) * grain_size;
  const int current_block_div = (current_block / num_chunks_3) * grain_size;
  const int offset1_tid2 = (current_block_mod) + tid2;
  const int offset2_tid2 = (current_block_div) + tid2;
  const int offset1_tid3 = (current_block_mod) + tid3;
  const int offset2_tid3 = (current_block_div) + tid3;
  const int ii3 = offset1_tid3;
  const int ii21 = offset2_tid3;
  if (ii3 < size3) {
#pragma unroll
    if (offset2_tid2 + 3 * 8 < size2) {
      for (int sub = 0; sub < 4; sub++) {
        const int ii2 = offset2_tid2 + sub * 8;
        const int ii = ii2 * size3 + ii3;
        tile[tid2 + sub * 8][tid3] = __ldg(reinterpret_cast<const __half*>(input) + offset + ii);
      }
    } else {
      for (int sub = 0; sub < 4; sub++) {
        const int ii2 = offset2_tid2 + sub * 8;
        if (ii2 < size2) {
          const int ii = ii2 * size3 + ii3;
          tile[tid2 + sub * 8][tid3] = __ldg(reinterpret_cast<const __half*>(input) + offset + ii);
        }
      }
    }
  }

  __syncthreads();

  if (ii21 < size2) {
#pragma unroll
    for (int sub = 0; sub < 4; sub++) {
      const int ii31 = offset1_tid2 + sub * 8;
      if (ii31 < size3) {
        const int ii1 = ii21 * size3 + ii31;
        const int j = (ii1 % size3) * size2;
        const int i = (ii1 / size3);
        output[offset + j + i] = tile[tid3][tid2 + sub * 8];
      }
    }
  }
}

void transpose_kernelLauncher(
    c10::Half* input, // [batch_size x None]
    c10::Half* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    const int* block_offsets,
    const int* offsets,
    const int batch_size,
    const int block_numel,
    const int* size_dim2,
    const int* size_dim3,
    const hipStream_t stream)
{
  dim3 grid;
  grid.x = block_numel,

  transpose<32><<<grid, 256, 0, stream>>>(
      input,
      output,
      block_offsets,
      offsets,
      batch_size,
      size_dim2,
      size_dim3);
}

}
} // namespace nested_tensor
